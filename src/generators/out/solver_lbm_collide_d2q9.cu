#include "hip/hip_runtime.h"
#include "solver.h"

#if (N_Q==9)

__global__
void Cu_Collide_d2q9
(
	int n_ids_idev_L, int *id_set_idev_L, long int n_maxcells, ufloat_t dx_L, ufloat_t tau_L, ufloat_t tau_ratio,
	int *cblock_ID_onb, int *cblock_ID_nbr, int *cblock_ID_nbr_child, int *cblock_ID_mask, int n_maxcblocks,
	int *cells_ID_mask, ufloat_t *cells_f_F
)
{
	__shared__ int s_ID_cblock[M_CBLOCK];
#if (B_TYPE==1||S_LES==1)
	__shared__ ufloat_t s_u[(Nbx+2)*(Nbx+2)];
	__shared__ ufloat_t s_v[(Nbx+2)*(Nbx+2)];
#endif
	int kap = blockIdx.x*blockDim.x + threadIdx.x;
	int block_on_boundary = 0;
	int i_kap = -1;
	int i_kap_bc = -1;
	int I_kap = threadIdx.x % Nbx;
	int J_kap = (threadIdx.x / Nbx) % Nbx;
	ufloat_t f_0 = N_Pf(0.0);
	ufloat_t f_1 = N_Pf(0.0);
	ufloat_t f_2 = N_Pf(0.0);
	ufloat_t f_3 = N_Pf(0.0);
	ufloat_t f_4 = N_Pf(0.0);
	ufloat_t f_5 = N_Pf(0.0);
	ufloat_t f_6 = N_Pf(0.0);
	ufloat_t f_7 = N_Pf(0.0);
	ufloat_t f_8 = N_Pf(0.0);
	ufloat_t rho_kap = N_Pf(0.0);
	ufloat_t u_kap = N_Pf(0.0);
	ufloat_t v_kap = N_Pf(0.0);
	ufloat_t tmp_i = N_Pf(0.0);
#if (S_LES==1)
	ufloat_t tmp_j = N_Pf(0.0);
	ufloat_t tmp_k = N_Pf(0.0);
#endif
	ufloat_t cdotu = N_Pf(0.0);
	ufloat_t udotu = N_Pf(0.0);
	ufloat_t omeg = dx_L / tau_L;
	ufloat_t omegp = N_Pf(1.0) - omeg;

	s_ID_cblock[threadIdx.x] = -1;
	if (kap < n_ids_idev_L)
	{
		i_kap = id_set_idev_L[kap];
		s_ID_cblock[threadIdx.x] = i_kap;
	}
	__syncthreads();

	// Loop over block Ids.
	for (int k = 0; k < M_CBLOCK; k++)
	{
		int i_kap_b = s_ID_cblock[k];
		int nbr_kap_b = -1;
		i_kap_bc = -1;
		block_on_boundary = 0;

		if (i_kap_b > -1)
		{
			i_kap_bc = cblock_ID_nbr_child[i_kap_b];
			block_on_boundary = cblock_ID_mask[i_kap_b];
		}

		if ( i_kap_b > -1 && (i_kap_bc < 0 || block_on_boundary == 1) )
		{
			// Load DDFs and compute macroscopic properties.
			f_0 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 0*n_maxcells];
			f_1 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 3*n_maxcells];
			f_2 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 4*n_maxcells];
			f_3 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 1*n_maxcells];
			f_4 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 2*n_maxcells];
			f_5 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 7*n_maxcells];
			f_6 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 8*n_maxcells];
			f_7 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 5*n_maxcells];
			f_8 = cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 6*n_maxcells];
			rho_kap = +f_0 +f_1 +f_2 +f_3 +f_4 +f_5 +f_6 +f_7 +f_8;
			u_kap = ( +f_1 -f_3 +f_5 -f_6 -f_7 +f_8) / rho_kap;
			v_kap = ( +f_2 -f_4 +f_5 +f_6 -f_7 -f_8) / rho_kap;
			udotu = u_kap*u_kap + v_kap*v_kap;
#if (B_TYPE==1||S_LES==1)
			s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)] = u_kap;
			s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)] = v_kap;
			__syncthreads();
#endif

			// Get turublent viscosity for Large Eddy Simulation.
#if (S_LES==1)
			// Extrapolate macroscopic properties to block edges.
			if (I_kap==0)
			{
				s_u[0+(Nbx+2)*(J_kap+1)] = 4*s_u[1+(Nbx+2)*(J_kap+1)] - 6*s_u[2+(Nbx+2)*(J_kap+1)] + 4*s_u[3+(Nbx+2)*(J_kap+1)] - s_u[4+(Nbx+2)*(J_kap+1)];
				s_v[0+(Nbx+2)*(J_kap+1)] = 4*s_v[1+(Nbx+2)*(J_kap+1)] - 6*s_v[2+(Nbx+2)*(J_kap+1)] + 4*s_v[3+(Nbx+2)*(J_kap+1)] - s_v[4+(Nbx+2)*(J_kap+1)];
				s_u[5+(Nbx+2)*(J_kap+1)] = 4*s_u[4+(Nbx+2)*(J_kap+1)] - 6*s_u[3+(Nbx+2)*(J_kap+1)] + 4*s_u[2+(Nbx+2)*(J_kap+1)] - s_u[1+(Nbx+2)*(J_kap+1)];
				s_v[5+(Nbx+2)*(J_kap+1)] = 4*s_v[4+(Nbx+2)*(J_kap+1)] - 6*s_v[3+(Nbx+2)*(J_kap+1)] + 4*s_v[2+(Nbx+2)*(J_kap+1)] - s_v[1+(Nbx+2)*(J_kap+1)];
			}
			if (J_kap==0)
			{
				s_u[(I_kap+1)+(Nbx+2)*(0)] = 4*s_u[(I_kap+1)+(Nbx+2)*(1)] - 6*s_u[(I_kap+1)+(Nbx+2)*(2)] + 4*s_u[(I_kap+1)+(Nbx+2)*(3)] - s_u[(I_kap+1)+(Nbx+2)*(4)];
				s_v[(I_kap+1)+(Nbx+2)*(0)] = 4*s_v[(I_kap+1)+(Nbx+2)*(1)] - 6*s_v[(I_kap+1)+(Nbx+2)*(2)] + 4*s_v[(I_kap+1)+(Nbx+2)*(3)] - s_v[(I_kap+1)+(Nbx+2)*(4)];
				s_u[(I_kap+1)+(Nbx+2)*(5)] = 4*s_u[(I_kap+1)+(Nbx+2)*(4)] - 6*s_u[(I_kap+1)+(Nbx+2)*(3)] + 4*s_u[(I_kap+1)+(Nbx+2)*(2)] - s_u[(I_kap+1)+(Nbx+2)*(1)];
				s_v[(I_kap+1)+(Nbx+2)*(5)] = 4*s_v[(I_kap+1)+(Nbx+2)*(4)] - 6*s_v[(I_kap+1)+(Nbx+2)*(3)] + 4*s_v[(I_kap+1)+(Nbx+2)*(2)] - s_v[(I_kap+1)+(Nbx+2)*(1)];
			}
			__syncthreads();

			// Compute turbulent viscosity, storing S_{ij}^d in tmp_j, S_{ij} in tmp_k.
			tmp_i = N_Pf(0.0);
			tmp_j = N_Pf(0.0);
			tmp_k = N_Pf(0.0);
				// (11,22)
			tmp_i += (( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)])*(s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)]))*( + (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])));
				// (12,12)
			tmp_j += (( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)]))*( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])));
			tmp_j = N_Pf(0.25)*(tmp_i - tmp_j);

			// Denominator.
			tmp_k += (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)])*(s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)]);
			tmp_k += (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)]);
			tmp_k += (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)]);
			tmp_k += (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)]);

			// Compute t_eff.
			tmp_k = tmp_k/(N_Pf(4.0)*dx_L*dx_L);
			tmp_i = (N_Pf(0.070000000000000))*sqrt(tmp_j/tmp_k);
			if (isnan(tmp_i))
			{
				tmp_i = N_Pf(0.0);
			}
			omeg = dx_L / (   N_Pf(3.0)*(v0 + tmp_i) + N_Pf(0.5)*dx_L   );
			omegp = N_Pf(1.0) - omeg;
			tau_ratio = N_Pf(0.25) + (N_Pf(0.75)*tau_L - N_Pf(0.25)*dx_L)*(omeg/dx_L);
#endif

			// Collision step.
			cdotu = N_Pf(0.0);
			f_0 = f_0*omegp + ( N_Pf(0.444444444444444)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = +u_kap;
			f_1 = f_1*omegp + ( N_Pf(0.111111111111111)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = +v_kap;
			f_2 = f_2*omegp + ( N_Pf(0.111111111111111)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = -u_kap;
			f_3 = f_3*omegp + ( N_Pf(0.111111111111111)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = -v_kap;
			f_4 = f_4*omegp + ( N_Pf(0.111111111111111)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = +u_kap+v_kap;
			f_5 = f_5*omegp + ( N_Pf(0.027777777777778)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = -u_kap+v_kap;
			f_6 = f_6*omegp + ( N_Pf(0.027777777777778)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = -u_kap-v_kap;
			f_7 = f_7*omegp + ( N_Pf(0.027777777777778)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;
			cdotu = +u_kap-v_kap;
			f_8 = f_8*omegp + ( N_Pf(0.027777777777778)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu) )*omeg;

			// Impose boundary conditions.
			block_on_boundary = cblock_ID_onb[i_kap_b];
			if (block_on_boundary == 1)
			{
				// nbr 1
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 1*n_maxcblocks];
					// p = 1
				if ((I_kap+1==Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_1 = f_1 - N_Pf(2.0)*N_Pf(0.111111111111111)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 5
				if ((I_kap+1==Nbx) && (J_kap+1< Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_5 = f_5 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 8
				if ((I_kap+1==Nbx) && (J_kap-1>= 0))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_8 = f_8 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 2
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 2*n_maxcblocks];
					// p = 2
				if ((J_kap+1==Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.000000000000000);
						f_2 = f_2 - N_Pf(2.0)*N_Pf(0.111111111111111)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 5
				if ((I_kap+1< Nbx) && (J_kap+1==Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_5 = f_5 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 6
				if ((I_kap-1>= 0) && (J_kap+1==Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_6 = f_6 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 3
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 3*n_maxcblocks];
					// p = 3
				if ((I_kap-1==-1))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_3 = f_3 - N_Pf(2.0)*N_Pf(0.111111111111111)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 6
				if ((I_kap-1==-1) && (J_kap+1< Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_6 = f_6 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 7
				if ((I_kap-1==-1) && (J_kap-1>= 0))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_7 = f_7 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 4
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 4*n_maxcblocks];
					// p = 4
				if ((J_kap-1==-1))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.000000000000000);
						f_4 = f_4 - N_Pf(2.0)*N_Pf(0.111111111111111)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 7
				if ((I_kap-1>= 0) && (J_kap-1==-1))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_7 = f_7 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
					// p = 8
				if ((I_kap+1< Nbx) && (J_kap-1==-1))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_8 = f_8 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 5
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 5*n_maxcblocks];
					// p = 5
				if ((I_kap+1==Nbx) && (J_kap+1==Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_5 = f_5 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 6
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 6*n_maxcblocks];
					// p = 6
				if ((I_kap-1==-1) && (J_kap+1==Nbx))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_6 = f_6 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 7
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 7*n_maxcblocks];
					// p = 7
				if ((I_kap-1==-1) && (J_kap-1==-1))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(-0.050000000000000);
						f_7 = f_7 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
				// nbr 8
				nbr_kap_b = cblock_ID_nbr[i_kap_b + 8*n_maxcblocks];
					// p = 8
				if ((I_kap+1==Nbx) && (J_kap-1==-1))
				{
					if (nbr_kap_b == -4)
					{
						cdotu = N_Pf(0.050000000000000);
						f_8 = f_8 - N_Pf(2.0)*N_Pf(0.027777777777778)*N_Pf(3.0)*cdotu;
					}
				}
			}

			// Write fi* to global memory.
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 0*n_maxcells] = f_0;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 1*n_maxcells] = f_1;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 2*n_maxcells] = f_2;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 3*n_maxcells] = f_3;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 4*n_maxcells] = f_4;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 5*n_maxcells] = f_5;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 6*n_maxcells] = f_6;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 7*n_maxcells] = f_7;
			cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 8*n_maxcells] = f_8;
			__syncthreads();
		}
	}
}


int Solver_LBM::S_Collide_d2q9(int i_dev, int L)
{
	if (mesh->n_ids[i_dev][L] > 0)
	{
		Cu_Collide_d2q9<<<(M_CBLOCK+mesh->n_ids[i_dev][L]-1)/M_CBLOCK,M_CBLOCK,0,mesh->streams[i_dev]>>>
		(
			mesh->n_ids[i_dev][L], mesh->c_id_set[i_dev][L], mesh->n_maxcells, dx_vec[L], tau_vec[L], tau_ratio_vec_C2F[L],
			mesh->c_cblock_ID_onb[i_dev], mesh->c_cblock_ID_nbr[i_dev], mesh->c_cblock_ID_nbr_child[i_dev], mesh->c_cblock_ID_mask[i_dev], mesh->n_maxcblocks,
			mesh->c_cells_ID_mask[i_dev], mesh->c_cells_f_F[i_dev]
		);
	}

	return 0;
}

#endif

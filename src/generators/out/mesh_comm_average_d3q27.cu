#include "hip/hip_runtime.h"
#include "mesh.h"

#if (N_Q==27)

template <int ave_type = 0>
__global__
void Cu_Average_d3q27
(
	int n_ids_idev_L, int *id_set_idev_L, long int n_maxcells, ufloat_t dx_Lm1, ufloat_t dx_L, ufloat_t tau_L, ufloat_t tau_ratio,
	int *cblock_ID_onb, int *cblock_ID_mask, int *cblock_ID_nbr_child, int n_maxcblocks,
	int *cells_ID_mask, ufloat_t *cells_f_F
)
{
	__shared__ int s_ID_cblock[M_CBLOCK];
	__shared__ int s_ID_mask_child[M_CBLOCK];
	__shared__ ufloat_t s_Fc[M_CBLOCK];
#if (S_LES==1)
	__shared__ ufloat_t s_Feq[M_CBLOCK];
	__shared__ ufloat_t s_tau[M_CBLOCK];
#endif
#if (B_TYPE==1||S_LES==1)
	__shared__ ufloat_t s_u[(Nbx+2)*(Nbx+2)*(Nbx+2)];
	__shared__ ufloat_t s_v[(Nbx+2)*(Nbx+2)*(Nbx+2)];
	__shared__ ufloat_t s_w[(Nbx+2)*(Nbx+2)*(Nbx+2)];
#endif
	int kap = blockIdx.x*blockDim.x + threadIdx.x;
	int block_on_boundary = 0;
	int i_kap = -1;
	int i_kap_bc = -1;
	int I_kap = threadIdx.x % Nbx;
	int J_kap = (threadIdx.x / Nbx) % Nbx;
	int K_kap = (threadIdx.x / Nbx) / Nbx;
	int child0_IJK = 2*((threadIdx.x % Nbx)%2) + Nbx*(2*(((threadIdx.x / Nbx) % Nbx)%2)) + Nbx*Nbx*(2*(((threadIdx.x / Nbx) / Nbx)%2));
	ufloat_t f_0 = N_Pf(0.0);
	ufloat_t f_1 = N_Pf(0.0);
	ufloat_t f_2 = N_Pf(0.0);
	ufloat_t f_3 = N_Pf(0.0);
	ufloat_t f_4 = N_Pf(0.0);
	ufloat_t f_5 = N_Pf(0.0);
	ufloat_t f_6 = N_Pf(0.0);
	ufloat_t f_7 = N_Pf(0.0);
	ufloat_t f_8 = N_Pf(0.0);
	ufloat_t f_9 = N_Pf(0.0);
	ufloat_t f_10 = N_Pf(0.0);
	ufloat_t f_11 = N_Pf(0.0);
	ufloat_t f_12 = N_Pf(0.0);
	ufloat_t f_13 = N_Pf(0.0);
	ufloat_t f_14 = N_Pf(0.0);
	ufloat_t f_15 = N_Pf(0.0);
	ufloat_t f_16 = N_Pf(0.0);
	ufloat_t f_17 = N_Pf(0.0);
	ufloat_t f_18 = N_Pf(0.0);
	ufloat_t f_19 = N_Pf(0.0);
	ufloat_t f_20 = N_Pf(0.0);
	ufloat_t f_21 = N_Pf(0.0);
	ufloat_t f_22 = N_Pf(0.0);
	ufloat_t f_23 = N_Pf(0.0);
	ufloat_t f_24 = N_Pf(0.0);
	ufloat_t f_25 = N_Pf(0.0);
	ufloat_t f_26 = N_Pf(0.0);
	ufloat_t rho_kap = N_Pf(0.0);
	ufloat_t u_kap = N_Pf(0.0);
	ufloat_t v_kap = N_Pf(0.0);
	ufloat_t w_kap = N_Pf(0.0);
	ufloat_t tmp_i = N_Pf(0.0);
#if (S_LES==1)
	ufloat_t tmp_j = N_Pf(0.0);
	ufloat_t tmp_k = N_Pf(0.0);
#endif
	ufloat_t cdotu = N_Pf(0.0);
	ufloat_t udotu = N_Pf(0.0);

	s_ID_cblock[threadIdx.x] = -1;
	if (kap < n_ids_idev_L)
	{
		i_kap = id_set_idev_L[kap];
		s_ID_cblock[threadIdx.x] = i_kap;
	}
	__syncthreads();

	// Loop over block Ids.
	for (int k = 0; k < M_CBLOCK; k++)
	{
		int i_kap_b = s_ID_cblock[k];
		i_kap_bc = -1;
		block_on_boundary = 0;

		if (i_kap_b > -1)
		{
			i_kap_bc = cblock_ID_nbr_child[i_kap_b];
			block_on_boundary = cblock_ID_mask[i_kap_b];
		}

		if ( i_kap_bc > -1 && (ave_type == 2 || block_on_boundary == 1) )
		{
			for (int xc_k = 0; xc_k < 2; xc_k++)
			{
				for (int xc_j = 0; xc_j < 2; xc_j++)
				{
					for (int xc_i = 0; xc_i < 2; xc_i++)
					{
						int xc = xc_i + 2*xc_j + 4*xc_k;

						// Load DDFs and compute macroscopic properties.
						f_0 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 0*n_maxcells];
						f_1 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 2*n_maxcells];
						f_2 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 1*n_maxcells];
						f_3 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 4*n_maxcells];
						f_4 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 3*n_maxcells];
						f_5 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 6*n_maxcells];
						f_6 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 5*n_maxcells];
						f_7 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 8*n_maxcells];
						f_8 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 7*n_maxcells];
						f_9 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 10*n_maxcells];
						f_10 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 9*n_maxcells];
						f_11 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 12*n_maxcells];
						f_12 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 11*n_maxcells];
						f_13 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 14*n_maxcells];
						f_14 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 13*n_maxcells];
						f_15 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 16*n_maxcells];
						f_16 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 15*n_maxcells];
						f_17 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 18*n_maxcells];
						f_18 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 17*n_maxcells];
						f_19 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 20*n_maxcells];
						f_20 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 19*n_maxcells];
						f_21 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 22*n_maxcells];
						f_22 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 21*n_maxcells];
						f_23 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 24*n_maxcells];
						f_24 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 23*n_maxcells];
						f_25 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 26*n_maxcells];
						f_26 = cells_f_F[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x + 25*n_maxcells];
						rho_kap = +f_0 +f_1 +f_2 +f_3 +f_4 +f_5 +f_6 +f_7 +f_8 +f_9 +f_10 +f_11 +f_12 +f_13 +f_14 +f_15 +f_16 +f_17 +f_18 +f_19 +f_20 +f_21 +f_22 +f_23 +f_24 +f_25 +f_26;
						u_kap = ( +f_1 -f_2 +f_7 -f_8 +f_9 -f_10 +f_13 -f_14 +f_15 -f_16 +f_19 -f_20 +f_21 -f_22 +f_23 -f_24 -f_25 +f_26) / rho_kap;
						v_kap = ( +f_3 -f_4 +f_7 -f_8 +f_11 -f_12 -f_13 +f_14 +f_17 -f_18 +f_19 -f_20 +f_21 -f_22 -f_23 +f_24 +f_25 -f_26) / rho_kap;
						w_kap = ( +f_5 -f_6 +f_9 -f_10 +f_11 -f_12 -f_15 +f_16 -f_17 +f_18 +f_19 -f_20 -f_21 +f_22 +f_23 -f_24 +f_25 -f_26) / rho_kap;
						udotu = u_kap*u_kap + v_kap*v_kap + w_kap*w_kap;
#if (B_TYPE==1||S_LES==1)
						s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = u_kap;
						s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = v_kap;
						s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = w_kap;
						__syncthreads();
#endif

						// Get turublent viscosity for Large Eddy Simulation.
#if (S_LES==1)
						// Extrapolate macroscopic properties to block edges.
						if (I_kap==0)
						{
							s_u[0+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_u[1+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_u[2+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_u[3+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[4+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_v[0+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_v[1+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_v[2+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_v[3+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[4+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_w[0+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_w[1+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_w[2+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_w[3+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[4+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_u[5+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_u[4+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_u[3+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_u[2+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[1+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_v[5+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_v[4+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_v[3+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_v[2+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[1+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_w[5+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_w[4+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_w[3+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_w[2+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[1+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
						}
						if (J_kap==0)
						{
							s_u[(I_kap+1)+(Nbx+2)*(0)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_u[(I_kap+1)+(Nbx+2)*(1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_u[(I_kap+1)+(Nbx+2)*(2)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_u[(I_kap+1)+(Nbx+2)*(3)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(4)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_v[(I_kap+1)+(Nbx+2)*(0)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_v[(I_kap+1)+(Nbx+2)*(1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_v[(I_kap+1)+(Nbx+2)*(2)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_v[(I_kap+1)+(Nbx+2)*(3)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(4)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_w[(I_kap+1)+(Nbx+2)*(0)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_w[(I_kap+1)+(Nbx+2)*(1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_w[(I_kap+1)+(Nbx+2)*(2)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_w[(I_kap+1)+(Nbx+2)*(3)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(4)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_u[(I_kap+1)+(Nbx+2)*(5)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_u[(I_kap+1)+(Nbx+2)*(4)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_u[(I_kap+1)+(Nbx+2)*(3)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_u[(I_kap+1)+(Nbx+2)*(2)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_v[(I_kap+1)+(Nbx+2)*(5)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_v[(I_kap+1)+(Nbx+2)*(4)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_v[(I_kap+1)+(Nbx+2)*(3)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_v[(I_kap+1)+(Nbx+2)*(2)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
							s_w[(I_kap+1)+(Nbx+2)*(5)+(Nbx+2)*(Nbx+2)*(K_kap+1)] = 4*s_w[(I_kap+1)+(Nbx+2)*(4)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - 6*s_w[(I_kap+1)+(Nbx+2)*(3)+(Nbx+2)*(Nbx+2)*(K_kap+1)] + 4*s_w[(I_kap+1)+(Nbx+2)*(2)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(1)+(Nbx+2)*(Nbx+2)*(K_kap+1)];
						}
						if (K_kap==0)
						{
							s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(0)] = 4*s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(1)] - 6*s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(2)] + 4*s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(3)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(4)];
							s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(0)] = 4*s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(1)] - 6*s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(2)] + 4*s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(3)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(4)];
							s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(0)] = 4*s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(1)] - 6*s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(2)] + 4*s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(3)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(4)];
							s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(5)] = 4*s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(4)] - 6*s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(3)] + 4*s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(2)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(1)];
							s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(5)] = 4*s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(4)] - 6*s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(3)] + 4*s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(2)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(1)];
							s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(5)] = 4*s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(4)] - 6*s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(3)] + 4*s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(2)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(1)];
						}
						__syncthreads();

						// Compute turbulent viscosity, storing S_{ij}^d in tmp_j, S_{ij} in tmp_k.
						tmp_i = N_Pf(0.0);
						tmp_j = N_Pf(0.0);
						tmp_k = N_Pf(0.0);
							// (11,22)
						tmp_i += (( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]))*( + (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])));
							// (11,33)
						tmp_i += (( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]))*( + (s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])));
							// (22,33)
						tmp_i += (( + (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]))*( + (s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])));
							// (12,12)
						tmp_j += (( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]))*( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])));
							// (13,13)
						tmp_j += (( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]))*( + (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])));
							// (23,23)
						tmp_j += (( + (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]))*( + (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]) + (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])));
						tmp_j = N_Pf(0.25)*(tmp_i - tmp_j);

						// Denominator.
						tmp_k += (s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_u[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]);
						tmp_k += (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]);
						tmp_k += (s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_u[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]);
						tmp_k += (s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]);
						tmp_k += (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]);
						tmp_k += (s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_v[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]);
						tmp_k += (s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap-1+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]);
						tmp_k += (s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap-1+1)+(Nbx+2)*(Nbx+2)*(K_kap+1)]);
						tmp_k += (s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)])*(s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap+1+1)] - s_w[(I_kap+1)+(Nbx+2)*(J_kap+1)+(Nbx+2)*(Nbx+2)*(K_kap-1+1)]);

						// Compute t_eff.
						tmp_k = tmp_k/(N_Pf(4.0)*dx_L*dx_L);
						tmp_i = (N_Pf(0.070000000000000))*sqrt(tmp_j/tmp_k);
						if (isnan(tmp_i))
						{
							tmp_i = N_Pf(0.0);
						}
						s_tau[threadIdx.x] = N_Pf(3.0)*(v0 + tmp_i) + N_Pf(0.5)*dx_L;
#endif

						// Average rescaled fi to parent if applicable.
						s_ID_mask_child[threadIdx.x] = cells_ID_mask[(i_kap_bc + xc)*M_CBLOCK + threadIdx.x];
						if (ave_type > 0 && s_ID_mask_child[threadIdx.x] < 2)
						{
							s_ID_mask_child[threadIdx.x] = 1;
						}

							// p = 0
						cdotu = N_Pf(0.0);
						tmp_i = N_Pf(0.296296296296296)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_0 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_0;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 0*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 0*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 1
						cdotu = +u_kap;
						tmp_i = N_Pf(0.074074074074074)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_1 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_1;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 2*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 2*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 2
						cdotu = -u_kap;
						tmp_i = N_Pf(0.074074074074074)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_2 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_2;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 1*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 1*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 3
						cdotu = +v_kap;
						tmp_i = N_Pf(0.074074074074074)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_3 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_3;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 4*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 4*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 4
						cdotu = -v_kap;
						tmp_i = N_Pf(0.074074074074074)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_4 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_4;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 3*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 3*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 5
						cdotu = +w_kap;
						tmp_i = N_Pf(0.074074074074074)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_5 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_5;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 6*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 6*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 6
						cdotu = -w_kap;
						tmp_i = N_Pf(0.074074074074074)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_6 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_6;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 5*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 5*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 7
						cdotu = +u_kap+v_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_7 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_7;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 8*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 8*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 8
						cdotu = -u_kap-v_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_8 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_8;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 7*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 7*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 9
						cdotu = +u_kap+w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_9 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_9;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 10*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 10*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 10
						cdotu = -u_kap-w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_10 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_10;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 9*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 9*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 11
						cdotu = +v_kap+w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_11 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_11;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 12*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 12*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 12
						cdotu = -v_kap-w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_12 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_12;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 11*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 11*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 13
						cdotu = +u_kap-v_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_13 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_13;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 14*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 14*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 14
						cdotu = -u_kap+v_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_14 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_14;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 13*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 13*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 15
						cdotu = +u_kap-w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_15 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_15;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 16*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 16*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 16
						cdotu = -u_kap+w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_16 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_16;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 15*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 15*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 17
						cdotu = +v_kap-w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_17 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_17;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 18*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 18*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 18
						cdotu = -v_kap+w_kap;
						tmp_i = N_Pf(0.018518518518519)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_18 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_18;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 17*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 17*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 19
						cdotu = +u_kap+v_kap+w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_19 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_19;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 20*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 20*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 20
						cdotu = -u_kap-v_kap-w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_20 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_20;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 19*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 19*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 21
						cdotu = +u_kap+v_kap-w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_21 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_21;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 22*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 22*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 22
						cdotu = -u_kap-v_kap+w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_22 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_22;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 21*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 21*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 23
						cdotu = +u_kap-v_kap+w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_23 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_23;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 24*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 24*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 24
						cdotu = -u_kap+v_kap-w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_24 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_24;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 23*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 23*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 25
						cdotu = -u_kap+v_kap+w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_25 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_25;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 26*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 26*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();

							// p = 26
						cdotu = +u_kap-v_kap-w_kap;
						tmp_i = N_Pf(0.004629629629630)*rho_kap*(N_Pf(1.0) + N_Pf(3.0)*cdotu + N_Pf(4.5)*cdotu*cdotu - N_Pf(1.5)*udotu);
#if (S_LES==0)
						s_Fc[threadIdx.x] = tmp_i + (f_26 - tmp_i)*tau_ratio;
#else
						s_Fc[threadIdx.x] = f_26;
						s_Feq[threadIdx.x] = tmp_i;
#endif
						__syncthreads();
#if (S_LES==0)
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 25*n_maxcells] = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
						}
#else // Storing interpolated fi_eq, fi, tau_ratio in tmp_i, tmp_j and tmp_k, respectively.
						if (s_ID_mask_child[child0_IJK] == 1 && I_kap >= 2*xc_i && I_kap < 2+2*xc_i && J_kap >= 2*xc_j && J_kap < 2+2*xc_j && K_kap >= 2*xc_k && K_kap <= 2+2*xc_k)
						{
							tmp_i = N_Pf(0.125)*( s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Feq[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Feq[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_j = N_Pf(0.125)*( s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_Fc[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_Fc[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							tmp_k = N_Pf(0.125)*( s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*0)] +  s_tau[(child0_IJK + 0 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*0 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 0 + Nbx*1 + Nbx*Nbx*1)] +  s_tau[(child0_IJK + 1 + Nbx*1 + Nbx*Nbx*1)] );
							cells_f_F[i_kap_b*M_CBLOCK + threadIdx.x + 25*n_maxcells] = tmp_i + (tmp_j - tmp_i)*( N_Pf(1.0) + N_Pf(0.25)*dx_Lm1/tmp_k );
						}
#endif
						__syncthreads();
					}
				}
			}
		}
	}
}


int Mesh::M_Average_d3q27(int i_dev, int L, int var, ufloat_t Cscale, ufloat_t Cscale2)
{
	if (n_ids[i_dev][L] > 0 && var == V_AVERAGE_INTERFACE)
	{
		Cu_Average_d3q27<0><<<(M_CBLOCK+n_ids[i_dev][L]-1)/M_CBLOCK,M_CBLOCK,0,streams[i_dev]>>>
		(
			n_ids[i_dev][L], c_id_set[i_dev][L], n_maxcells, dxf_vec[L+1], dxf_vec[L], Cscale, Cscale2,
			c_cblock_ID_onb[i_dev], c_cblock_ID_mask[i_dev], c_cblock_ID_nbr_child[i_dev], n_maxcblocks,
			c_cells_ID_mask[i_dev], c_cells_f_F[i_dev]
		);
	}
	if (n_ids[i_dev][L] > 0 && var == V_AVERAGE_BLOCK)
	{
		Cu_Average_d3q27<1><<<(M_CBLOCK+n_ids[i_dev][L]-1)/M_CBLOCK,M_CBLOCK,0,streams[i_dev]>>>
		(
			n_ids[i_dev][L], c_id_set[i_dev][L], n_maxcells, dxf_vec[L+1], dxf_vec[L], Cscale, Cscale2,
			c_cblock_ID_onb[i_dev], c_cblock_ID_mask[i_dev], c_cblock_ID_nbr_child[i_dev], n_maxcblocks,
			c_cells_ID_mask[i_dev], c_cells_f_F[i_dev]
		);
	}
	if (n_ids[i_dev][L] > 0 && var == V_AVERAGE_GRID)
	{
		Cu_Average_d3q27<2><<<(M_CBLOCK+n_ids[i_dev][L]-1)/M_CBLOCK,M_CBLOCK,0,streams[i_dev]>>>
		(
			n_ids[i_dev][L], c_id_set[i_dev][L], n_maxcells, dxf_vec[L+1], dxf_vec[L], Cscale, Cscale2,
			c_cblock_ID_onb[i_dev], c_cblock_ID_mask[i_dev], c_cblock_ID_nbr_child[i_dev], n_maxcblocks,
			c_cells_ID_mask[i_dev], c_cells_f_F[i_dev]
		);
	}

	return 0;
}

#endif

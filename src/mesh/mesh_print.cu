#include "hip/hip_runtime.h"
/**************************************************************************************/
/*                                                                                    */
/*  Author: Khodr Jaber                                                               */
/*  Affiliation: Turbulence Research Lab, University of Toronto                       */
/*                                                                                    */
/**************************************************************************************/

#include "mesh.h"

int Mesh::M_Print_FillBlock(int i_dev, int *Is, int i_kap, int L, double dx_f, int *mult_f, int vol, int *Nxi_f, double *tmp_data)
{
	// (I,J,K) are the indices for the top parent. All child indices moving forward are built from these.
	int child_0 = cblock_ID_nbr_child[i_dev][i_kap];
	
	if (child_0 >= 0 && L+1 < N_PRINT_LEVELS) // Has children, keep traversing.
	{
#if (N_DIM==3)
		for (int xk = 0; xk < 2; xk++)
#else
		int xk = 0;
#endif
		{
			for (int xj = 0; xj < 2; xj++)
			{
				for (int xi = 0; xi < 2; xi++)
				{
					Is[L+1 + 0*N_PRINT_LEVELS] = xi;
					Is[L+1 + 1*N_PRINT_LEVELS] = xj;
					Is[L+1 + 2*N_PRINT_LEVELS] = xk;
					int xc = xi + 2*xj + 4*xk;
					M_Print_FillBlock(i_dev, Is, child_0+xc, L+1, dx_f, mult_f, vol, Nxi_f, tmp_data);
				}
			}
		}
	}
	else // No children, print here.
	{
		// Get the macroscopic properties for this block.
		double out_u_[M_CBLOCK*(6+1)]; 
		double out_yplus_[M_CBLOCK];
		M_ComputeProperties(i_dev, i_kap, dxf_vec[L], out_u_, out_yplus_);
		
		// Modify the cell values in the region defined by the leaf block.
#if (N_DIM==3)
		for (int k = 0; k < Nbx; k++)
#else
		int k = 0;
#endif
		{
			for (int j = 0; j < Nbx; j++)
			{
				for (int i = 0; i < Nbx; i++)
				{
					int Ip = 0;
					int Jp = 0;
					int Kp = 0;
					for (int l = 0; l < L+1; l++)
					{
						Ip += mult_f[l]*Nbx*Is[l + 0*N_PRINT_LEVELS];
						Jp += mult_f[l]*Nbx*Is[l + 1*N_PRINT_LEVELS];
						Kp += mult_f[l]*Nbx*Is[l + 2*N_PRINT_LEVELS];
					}
					
#if (N_DIM==3)
					for (int kk = 0; kk < mult_f[L]; kk++)
#else
					int kk = 0;
#endif
					{
						for (int jj = 0; jj < mult_f[L]; jj++)
						{
							for (int ii = 0; ii < mult_f[L]; ii++)
							{
								int kap_i = i + Nbx*j + Nbx*Nbx*k;
								int Ipp = Ip + i*mult_f[L] + ii;
								int Jpp = Jp + j*mult_f[L] + jj;
								int Kpp = Kp + k*mult_f[L] + kk;
								long int Id = Ipp + Nxi_f[0]*Jpp + Nxi_f[0]*Nxi_f[1]*Kpp;
								
								tmp_data[Id + 0*vol] = out_u_[kap_i + 0*M_CBLOCK];
								tmp_data[Id + 1*vol] = out_u_[kap_i + 1*M_CBLOCK];
								tmp_data[Id + 2*vol] = out_u_[kap_i + 2*M_CBLOCK];
								tmp_data[Id + 3*vol] = out_u_[kap_i + 3*M_CBLOCK];
								tmp_data[Id + 4*vol] = out_u_[kap_i + 4*M_CBLOCK];
								tmp_data[Id + 5*vol] = out_u_[kap_i + 5*M_CBLOCK];
								tmp_data[Id + 6*vol] = out_u_[kap_i + 6*M_CBLOCK];
								tmp_data[Id + 7*vol] = sqrt(out_u_[kap_i + 1*M_CBLOCK]*out_u_[kap_i + 1*M_CBLOCK] + out_u_[kap_i + 2*M_CBLOCK]*out_u_[kap_i + 2*M_CBLOCK] + out_u_[kap_i + 3*M_CBLOCK]*out_u_[kap_i + 3*M_CBLOCK]);
								tmp_data[Id + 8*vol] = sqrt(out_u_[kap_i + 4*M_CBLOCK]*out_u_[kap_i + 4*M_CBLOCK] + out_u_[kap_i + 5*M_CBLOCK]*out_u_[kap_i + 5*M_CBLOCK] + out_u_[kap_i + 6*M_CBLOCK]*out_u_[kap_i + 6*M_CBLOCK]);
								tmp_data[Id + 9*vol] = L;
								tmp_data[Id + 10*vol] = i_kap;
							}
						}
					}
				}
			}
		}
	}
		
	return 0;
}

int Mesh::M_Print(int i_dev, int iter)
{
	// Parameters.
		// Domain extents (w.r.t root grid, I_min <= I < I_max).
	int I_min = 0;
	int I_max = Nxi[0]/Nbx;
	int J_min = 0;
	int J_max = Nxi[1]/Nbx;
	int K_min = 0;
	int K_max = 1;
#if (N_DIM==3)
	K_min = 3*(Nxi[2]/Nbx)/4;
	K_max = Nxi[2]/Nbx;
#endif
		// Resolution multiplier.
	int mult = 1;
	double dx_f = dxf_vec[0];
	for (int L = 1; L < N_PRINT_LEVELS; L++) 
	{
		mult *= 2;
		dx_f *= 0.5;
	}
	int *mult_f = new int[N_PRINT_LEVELS];
	for (int L = 0; L < N_PRINT_LEVELS; L++)
		mult_f[L] = pow(2.0, (double)(N_PRINT_LEVELS-1-L));
		// Resolution array.
	int Nxi_f[3];
	Nxi_f[0] = (I_max-I_min)*Nbx;
	Nxi_f[1] = (J_max-J_min)*Nbx;
	Nxi_f[2] = (K_max-K_min)*Nbx;
	for (int d = 0; d < 3; d++)
		Nxi_f[d] = Nxi[d];
	for (int d = 0; d < N_DIM; d++)
		Nxi_f[d] *= mult;
	int vol = Nxi_f[0]*Nxi_f[1]*Nxi_f[2];
		// Cell data arrays.
	int n_data = 3+3+1+1+1+1+1;
	double *tmp_data = new double[n_data*vol];
	double *tmp_data_b = new double[n_data*vol];
	for (long int p = 0; p < n_data*vol; p++)
		tmp_data[p] = -1.0;
		// Density.
	vtkNew<vtkDoubleArray> cell_data_density;
	cell_data_density->SetName("Density");
	cell_data_density->SetNumberOfComponents(1);
	cell_data_density->SetNumberOfTuples(vol);
		// Velocity.
	vtkNew<vtkDoubleArray> cell_data_velocity;
	cell_data_velocity->SetName("Velocity");
	cell_data_velocity->SetNumberOfComponents(3);
	cell_data_velocity->SetNumberOfTuples(vol);
		// Vorticity.
	vtkNew<vtkDoubleArray> cell_data_vorticity;
	cell_data_vorticity->SetName("Vorticity");
	cell_data_vorticity->SetNumberOfComponents(3);
	cell_data_vorticity->SetNumberOfTuples(vol);
		// Velocity Magnitude.
	vtkNew<vtkDoubleArray> cell_data_velmag;
	cell_data_velmag->SetName("Velocity Magnitude");
	cell_data_velmag->SetNumberOfComponents(1);
	cell_data_velmag->SetNumberOfTuples(vol);
		// Vorticity Magnitude.
	vtkNew<vtkDoubleArray> cell_data_vortmag;
	cell_data_vortmag->SetName("Vorticity Magnitude");
	cell_data_vortmag->SetNumberOfComponents(1);
	cell_data_vortmag->SetNumberOfTuples(vol);
		// AMR Level.
	vtkNew<vtkDoubleArray> cell_data_level;
	cell_data_level->SetName("AMR Level");
	cell_data_level->SetNumberOfComponents(1);
	cell_data_level->SetNumberOfTuples(vol);
		// Block Id.
	vtkNew<vtkDoubleArray> cell_data_blockid;
	cell_data_blockid->SetName("Block Id");
	cell_data_blockid->SetNumberOfComponents(1);
	cell_data_blockid->SetNumberOfTuples(vol);
	
	// Traverse the grid and fill data arrays.
	std::cout << "[-] Traversing grid, computing properties..." << std::endl;
	#pragma omp parallel for
	for (int kap = 0; kap < n_ids[i_dev][0]; kap++)
	{
		int Is[N_PRINT_LEVELS*3];
		for (int Ld = 0; Ld < N_PRINT_LEVELS*3; Ld++) Is[Ld] = 0;
		
		Is[0*N_PRINT_LEVELS] = coarse_I[i_dev][kap] - I_min;
		Is[1*N_PRINT_LEVELS] = coarse_J[i_dev][kap] - J_min;
		Is[2*N_PRINT_LEVELS] = coarse_K[i_dev][kap] - K_min;
		
		//std::cout << kap << " | " << Is[0*N_PRINT_LEVELS] << ", " << Is[1*N_PRINT_LEVELS] << ", " << Is[2*N_PRINT_LEVELS] << "(" << I_min << "," << I_max << " | " << J_min << "," << J_max << " | " << K_min << "," << K_max << ")" << std::endl;
		
		if (Is[0*N_PRINT_LEVELS] >= 0 && Is[0*N_PRINT_LEVELS] < I_max-I_min && Is[1*N_PRINT_LEVELS] >= 0 && Is[1*N_PRINT_LEVELS] < J_max-J_min && Is[2*N_PRINT_LEVELS] >= 0 && Is[2*N_PRINT_LEVELS] < K_max-K_min)
			M_Print_FillBlock(i_dev, Is, kap, 0, dx_f, mult_f, vol, Nxi_f, tmp_data);
	}
	std::cout << "    Finished traversal..." << std::endl;
	
	// Smoothing.
	int n_smooths = mult*Nbx;
	if (n_smooths > 0)
		std::cout << "[-] Smoothing grid..." << std::endl;
	#pragma omp parallel for
	for (int kap = 0; kap < vol; kap++)
	{
		for (int p = 0; p < 7; p++)
			tmp_data_b[kap + p*vol] = tmp_data[kap + p*vol];
	}
	for (int i = 0; i < n_smooths; i++)
	{
		std::cout << "    Smoothing iteration " << i << "..." << std::endl;
#if (N_DIM==2)
		#pragma omp parallel for
		for (int J = 1; J < Nxi_f[1]-1; J++)
		{
			for (int I = 1; I < Nxi_f[0]-1; I++)
			{
				int kap = (I) + Nxi_f[0]*(J);
				for (int p = 0; p < 9; p++)
				{
					tmp_data_b[kap + p*vol] = (
						tmp_data[(I+1) + Nxi_f[0]*(J) + p*vol] +
						tmp_data[(I-1) + Nxi_f[0]*(J) + p*vol] +
						tmp_data[(I) + Nxi_f[0]*(J+1) + p*vol] +
						tmp_data[(I) + Nxi_f[0]*(J-1) + p*vol]
					)/4.0;
				}
			}
		}
#else
		#pragma omp parallel for
		for (int K = 1; K < Nxi_f[2]-1; K++)
		{
			for (int J = 1; J < Nxi_f[1]-1; J++)
			{
				for (int I = 1; I < Nxi_f[0]-1; I++)
				{
					int kap = (I) + Nxi_f[0]*(J) + Nxi_f[0]*Nxi_f[1]*(K);
					for (int p = 0; p < 9; p++)
					{
						tmp_data_b[kap + p*vol] = (
							tmp_data[(I+1) + Nxi_f[0]*(J) + Nxi_f[0]*Nxi_f[1]*(K) + p*vol] +
							tmp_data[(I-1) + Nxi_f[0]*(J) + Nxi_f[0]*Nxi_f[1]*(K) + p*vol] +
							tmp_data[(I) + Nxi_f[0]*(J+1) + Nxi_f[0]*Nxi_f[1]*(K) + p*vol] +
							tmp_data[(I) + Nxi_f[0]*(J-1) + Nxi_f[0]*Nxi_f[1]*(K) + p*vol] +
							tmp_data[(I) + Nxi_f[0]*(J) + Nxi_f[0]*Nxi_f[1]*(K+1) + p*vol] +
							tmp_data[(I) + Nxi_f[0]*(J) + Nxi_f[0]*Nxi_f[1]*(K-1) + p*vol]
						)/6.0;
					}
				}
			}
		}
#endif
		#pragma omp parallel for
		for (int kap = 0; kap < vol; kap++)
		{
			for (int p = 0; p < 7; p++)
				tmp_data[kap + p*vol] = tmp_data_b[kap + p*vol];
		}
	}
	if (n_smooths > 0)
		std::cout << "    Finished smoothing grid..." << std::endl;
	
	// Insert data in VTK arrays.
	std::cout << "[-] Inserting data in VTK pointers..." << std::endl;
	#pragma omp parallel for
	for (long int kap = 0; kap < vol; kap++)
	{
		cell_data_density->SetTuple1(kap,
			tmp_data[kap+ 0*vol]
		);
		cell_data_velocity->SetTuple3(kap,
			tmp_data[kap+ 1*vol],
			tmp_data[kap+ 2*vol],
			tmp_data[kap+ 3*vol]
		);
		cell_data_vorticity->SetTuple3(kap, 
			tmp_data[kap+ 4*vol],
			tmp_data[kap+ 5*vol],
			tmp_data[kap+ 6*vol]
		);
		cell_data_velmag->SetTuple1(kap, 
			tmp_data[kap+ 7*vol]
		);
		cell_data_vortmag->SetTuple1(kap, 
			tmp_data[kap+ 8*vol]
		);
		cell_data_level->SetTuple1(kap,
			tmp_data[kap+ 9*vol]
		);
		cell_data_blockid->SetTuple1(kap,
			tmp_data[kap+ 10*vol]
		);
	}
	std::cout << "    Finished inserting data in VTK pointers..." << std::endl;
	
	// Image data from uniform grid.
	std::cout << "[-] Creating uniform grid..." << std::endl;
		// Parameters and initialization.
	//double origin[3] = {0.0,0.0,0.0};
	double origin[3] = {I_min*dx_f*mult, J_min*dx_f*mult, K_min*dx_f*mult};
	double spacing[3] = {dx_f, dx_f, dx_f};
	vtkNew<vtkUniformGrid> grid;
	vtkNew<vtkCellDataToPointData> cell_to_points;
	vtkNew<vtkContourFilter> contour;
		// Set up image data grid.
	grid->Initialize();
	grid->SetOrigin(origin);
	grid->SetSpacing(spacing);
	grid->SetDimensions(Nxi_f[0]+1, Nxi_f[1]+1, N_DIM==2?1:Nxi_f[2]+1);
	grid->GetCellData()->AddArray(cell_data_density);
	grid->GetCellData()->AddArray(cell_data_velocity);
	grid->GetCellData()->AddArray(cell_data_vorticity);
	grid->GetCellData()->AddArray(cell_data_velmag);
	grid->GetCellData()->AddArray(cell_data_vortmag);
	grid->GetCellData()->AddArray(cell_data_level);
	grid->GetCellData()->AddArray(cell_data_blockid);
#if (N_CASE==1)
		// Blank invalid cells (these are identified by negative AMR level).
	grid->AllocateCellGhostArray();
	vtkUnsignedCharArray *ghosts = grid->GetCellGhostArray();
	#pragma omp parallel for
	for (long int kap = 0; kap < vol; kap++)
	{
		if (tmp_data[kap + 9*vol] < 0)
			ghosts->SetValue(kap, ghosts->GetValue(kap) | vtkDataSetAttributes::HIDDENCELL);
	}
#endif
	std::cout << "    Finished creating uniform grid..." << std::endl;
	
	// Image data processing.
	std::cout << "[-] Creating contours..." << std::endl;
		// Convert cell data to point data.
	cell_to_points->SetInputData(grid);
	cell_to_points->Update();
		// Contour for vorticity magnitude.
	cell_to_points->GetImageDataOutput()->GetPointData()->SetActiveScalars("Vorticity Magnitude");
	contour->SetInputConnection(0, cell_to_points->GetOutputPort(0));
	contour->SetNumberOfContours(3);
	contour->SetValue(0, 0.15);
	contour->SetValue(1, 0.2);
	contour->SetValue(2, 0.3);
	std::cout << "    Finished creating contours..." << std::endl;
	
	// Offscreen rendering.
	if ((iter+1)%P_RENDER == 0)
	{
		std::cout << "[-] Setting up renderer..." << std::endl;
			// Setup offscreen rendering.
		vtkNew<vtkNamedColors> colors;
		vtkNew<vtkGraphicsFactory> graphics_factory;
		graphics_factory->SetOffScreenOnlyMode(1);
		graphics_factory->SetUseMesaClasses(1);
			// Create mapper.
		vtkNew<vtkPolyDataMapper> mapper;
		mapper->SetInputConnection(contour->GetOutputPort(0));
			// Create actor.
		vtkNew<vtkActor> actor;
		actor->SetMapper(mapper);
		actor->GetProperty()->SetColor(colors->GetColor3d("White").GetData());
			// Create renderer.
		vtkNew<vtkRenderer> renderer;
		vtkNew<vtkRenderWindow> renderWindow;
		renderWindow->SetOffScreenRendering(1);
		renderWindow->AddRenderer(renderer);
			// Create camera.
		double cam_pos[3] = {1.8, -2.5, 1.25};
		//double cam_view_up[3] = {-0.066475, 0.21161, 0.975091};
		double cam_view_up[3] = {0.0, 0.0, 1.0};
		double cam_focal_point[3] = {0.5, 0.5, 0.5};
		vtkNew<vtkCamera> camera;
		renderer->SetActiveCamera(camera);
		camera->SetPosition(cam_pos);
		camera->SetViewUp(cam_view_up);
		camera->SetFocalPoint(cam_focal_point);
			// Add actor to scene and render.
		renderer->AddActor(actor);
		renderer->SetBackground(colors->GetColor3d("SlateGray").GetData());
		std::cout << "    Finished setup, rendering..." << std::endl;
		renderWindow->SetSize(1024, 1024);
		renderWindow->Render();
		std::cout << "    Rendered, taking photo..." << std::endl;
			// Print to PNG.
		vtkNew<vtkWindowToImageFilter> windowToImageFilter;
		windowToImageFilter->SetInput(renderWindow);
		windowToImageFilter->Update();
		vtkNew<vtkPNGWriter> photographer;
		size_t n_zeros = 7;
		std::string iter_string = std::to_string((iter+1)/P_RENDER);
		std::string padded_iter = std::string(n_zeros-std::min(n_zeros, iter_string.length()), '0') + iter_string;
		std::string photo_name = P_DIR_NAME + std::string("img/shot_") + padded_iter + ".png";
		photographer->SetFileName(photo_name.c_str());
		photographer->SetInputConnection(windowToImageFilter->GetOutputPort());
		photographer->Write();
		std::cout << "    Finished taking photo (no. " << (iter+1)/P_RENDER << ")..." << std::endl;
	}
	
	// Write grid.
	std::cout << "Finished building VTK dataset, writing..." << std::endl;
	std::string file_name = P_DIR_NAME + std::string("out_") + std::to_string(iter+1) + ".vti";
	vtkNew<vtkXMLImageDataWriter> writer;
	writer->SetInputData(cell_to_points->GetImageDataOutput());
	writer->SetFileName(file_name.c_str());
	writer->Write();
	std::cout << "Finished writing VTK dataset..." << std::endl;
	
	// Free allocations.
	delete[] mult_f;
	delete[] tmp_data;
	delete[] tmp_data_b;
	
	return 0;
}
